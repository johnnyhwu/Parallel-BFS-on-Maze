#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <queue>
#include "cell.hpp"


__global__ void bfsKernel(thrust::device_vector< thrust::device_vector<Cell> > & d_maze, float test) {
    int row = threadIdx.x;

    int maze_posY = d_maze[0][row].getRowPos;
    int maze_posX = d_maze[0][row].getColPos;

    printf("Thread: %d Position: %d %d\n", row, maze_posY, maze_posX);
}

void hostFE(vector< vector<Cell> > & h_maze, int height, int width) {
    // printf("height:%d  width:%d\n", height, width);
    // printf("maze[0][0]: %d", maze[0][0].getRowPos());

    queue<Cell> q;
    vector<bool> visited(height*width, false);

    // visit start point
    //maze[0][0].setFrom(Cell(0, 0));
    q.push(h_maze[0][0]);
    visited[0] = true;

    // Allocating device memory
    thrust::device_vector< thrust::device_vector<Cell> > d_maze = h_maze;


    dim3 dimBlock(4, 4);
    dim3 dimGrid(16/4, 16/4);


    bfsKernel<<<1, 5>>>(d_maze, 0);

}