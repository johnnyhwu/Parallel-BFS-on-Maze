#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <queue>
#include "cell.hpp"


__global__ void bfsKernel(int* current_queue, int* next_queue, int* current_queue_counter, int* next_queue_counter, \
                            int* d_node, int* d_edge, int* visited, int size, int num_edge, int destination, int* flag) {

    int index =  blockIdx.x * blockDim.x  + threadIdx.x; 
    
    if(index < *current_queue_counter) {
        int cell = current_queue[index];
        int n_added = 0;
        
        if(*flag == 0) {
            for(int i = d_node[cell]; i < d_node[cell+1]; i++) {
                int neighbor = d_edge[i];
                if(atomicCAS(visited+neighbor, 0, 1) == 0) {
                    if(neighbor == destination) {
                        atomicCAS(flag, 0, 1);  // Destination found
                        printf("Destination Found\n");
                        break;
                    }

                    int position = atomicAdd(next_queue_counter, 1);
                    next_queue[position] = neighbor;
                    n_added++;
                }
            }
        }
    }
}

void hostFE(int* node, int* edge, int num_edge, int height, int width) {

    int size = height * width;
    int* d_q1_counter;
    int* d_q2_counter;
    int* d_q1;
    int* d_q2;
    int* d_node;
    int* d_edge;
    int* d_visited;
    int* d_flag;
    int* q1_counter;
    int* q2_counter;
    int* counter_reset;
    int* flag;
    int destination = size-1;


    // Cuda memory allocation
    hipMalloc(&d_q1_counter, sizeof(int));
    hipMalloc(&d_q2_counter, sizeof(int));
    hipMalloc(&d_q1, sizeof(int) * size);
    hipMalloc(&d_q2, sizeof(int) * size);
    hipMalloc(&d_node, sizeof(int) * size);
    hipMalloc(&d_edge, sizeof(int) * num_edge);
    hipMalloc(&d_visited, sizeof(int) * size);
    hipMalloc(&d_flag, sizeof(int));
   

    // Insert node 0 into queue
    int* q_tmp;
    int* v_tmp;
    q_tmp = (int*)malloc(sizeof(int) * size);
    v_tmp = (int*)malloc(sizeof(int) * size);
    q1_counter = (int*)malloc(sizeof(int));
    q2_counter = (int*)malloc(sizeof(int));
    counter_reset = (int*)malloc(sizeof(int));
    hipHostAlloc(&counter_reset, sizeof(int), hipHostMallocDefault);
    flag = (int*)malloc(sizeof(int));
    for(int i = 0; i < size; i ++) {
        v_tmp[i] = 0;
    }
    q_tmp[0] = 0;
    v_tmp[0] = 1;
    *q1_counter = 1;
    *q2_counter = 0;
    *counter_reset = 0;
    *flag = 0;


    // Copy host memory to device memory
    hipMemcpy(d_q1, q_tmp, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_visited, v_tmp, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_q1_counter, q1_counter, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_q2_counter, q2_counter, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_node, node, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_edge, edge, sizeof(int) * num_edge, hipMemcpyHostToDevice);
    hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);
    

    // BFS algo, swaping between queue
    int level = 0;
    int thread_per_block = 1024;
    int block_per_grid = size/thread_per_block; 
    int* next_counter_ptr = (int*)malloc(sizeof(int));
    *next_counter_ptr = 1;
    printf("Size of matrix %d\n", size);

    while(*next_counter_ptr > 0) {
        if(level % 2 == 0) {

            hipMemcpy(d_q2_counter, counter_reset, sizeof(int), hipMemcpyHostToDevice);
            bfsKernel<<<block_per_grid, thread_per_block>>>(d_q1, d_q2, d_q1_counter, d_q2_counter, d_node, d_edge, d_visited, size, num_edge, destination, d_flag);
            hipMemcpy(next_counter_ptr, d_q2_counter, sizeof(int), hipMemcpyDeviceToHost);
        } else {

            hipMemcpy(d_q1_counter, counter_reset, sizeof(int), hipMemcpyHostToDevice);
            bfsKernel<<<block_per_grid, thread_per_block>>>(d_q2, d_q1, d_q2_counter, d_q1_counter, d_node, d_edge, d_visited, size, num_edge, destination, d_flag);
            hipMemcpy(next_counter_ptr, d_q1_counter, sizeof(int), hipMemcpyDeviceToHost);
        }
        level++;
    }


    // Free all allocated cuda memory
    hipFree(d_q1_counter);
    hipFree(d_q2_counter);
    hipFree(d_q1);
    hipFree(d_q2);
    hipFree(d_node);
    hipFree(d_edge);
    hipFree(d_visited);
    hipFree(d_flag);
}