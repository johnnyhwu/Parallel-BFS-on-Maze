#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void bfsKernel(int* d_output, float test) {
    int row = threadIdx.x;

    printf("%d %f fef\n", row, test);
    if (row == 0) {
        d_output[0] = 15;
    }
}

void hostFE (float test) {
    printf("%f\n", test);

    size_t size = sizeof(int)*5;

    int* d_output;
    int* h_output;
    hipMalloc(&d_output, size);
    h_output = (int*)malloc(size);

    dim3 dimBlock(4, 4);
    dim3 dimGrid(16/4, 16/4);
    bfsKernel<<<1, 5>>>(d_output, test);

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < 5; i++) {
        printf("%d ", h_output[i]);
    }

}